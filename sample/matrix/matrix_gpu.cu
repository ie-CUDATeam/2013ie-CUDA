#include "hip/hip_runtime.h"
#include <stdio.h>
#include <malloc.h>
#include <stdlib.h>
#include <time.h>
#include <cutil_inline.h>

#define MATRIX_SIZE 1024/*行列１辺の数*/
#define BLOCK_SIZE 16

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC);

int main(int argc, char** argv){
unsigned int matrixSize = sizeof(unsigned int) * MATRIX_SIZE * MATRIX_SIZE;

  int* hMatrixA;
  int* hMatrixB;
  int* hMatrixC;
  hMatrixA = (int*)malloc(matrixSize);
  hMatrixB = (int*)malloc(matrixSize);

/*初期値設定*/
  unsigned int col_idx, row_idx;
  for (col_idx = 0; col_idx < MATRIX_SIZE; col_idx++){
      for (row_idx = 0; row_idx < MATRIX_SIZE; row_idx++){
          hMatrixA[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024*1024);
          hMatrixB[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024*1024);
      }
  }

/*デバイス側の変数設定*/
  int* dMatrixA;
  int* dMatrixB;
  int* dMatrixC;
 
/*デバイスメモリ領域の確保*/
  cutilSafeCall(hipMalloc((void**)&dMatrixA, matrixSize));
  cutilSafeCall(hipMemcpy(dMatrixA, hMatrixA, matrixSize, hipMemcpyHostToDevice));
  cutilSafeCall(hipMalloc((void**)&dMatrixB, matrixSize));
  cutilSafeCall(hipMemcpy(dMatrixB, hMatrixB, matrixSize, hipMemcpyHostToDevice));
  cutilSafeCall(hipMalloc((void**)&dMatrixC, matrixSize));

/*ブロックサイズとグリッドサイズの設定*/
  dim3 block(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(MATRIX_SIZE/BLOCK_SIZE, MATRIX_SIZE/BLOCK_SIZE);

/*タイマーを作成して計測開始*/
  unsigned int timer = 0;
  CUT_SAFE_CALL( cutCreateTimer( &timer));
  CUT_SAFE_CALL( cutStartTimer( timer));

/*カーネルの起動*/
  matrixMul<<<grid, block>>>(dMatrixA, dMatrixB, dMatrixC);
  hipDeviceSynchronize();

/*結果の領域確保とデバイス側からのメモリ転送*/
  hMatrixC = (int*)malloc(matrixSize);
  cutilSafeCall(hipMemcpy(hMatrixC, dMatrixC, matrixSize, hipMemcpyDeviceToHost));

/*タイマーを停止しかかった時間を表示*/
  CUT_SAFE_CALL( cutStopTimer( timer));
  printf("Processing time: %f (msec)\n", cutGetTimerValue( timer));
  CUT_SAFE_CALL( cutDeleteTimer( timer));

/*ホスト・デバイスメモリの開放*/
  free(hMatrixA);
  free(hMatrixB);
  free(hMatrixC);
  cutilSafeCall(hipFree(dMatrixA));
  cutilSafeCall(hipFree(dMatrixB));
  cutilSafeCall(hipFree(dMatrixC));
 
/*終了処理*/
  hipDeviceReset();
  cutilExit(argc, argv);
 }
 
__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC){
  unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int scan_idx;
  unsigned int target = 0;

/*行列の演算を行う*/
 for (scan_idx = 0; scan_idx < MATRIX_SIZE; scan_idx++) {
   target +=inMatrixA[col_idx * MATRIX_SIZE + scan_idx] * inMatrixB[scan_idx * MATRIX_SIZE + row_idx];
   __syncthreads();
 }
 inMatrixC[col_idx * MATRIX_SIZE + row_idx] = target;
}
